#include "hip/hip_runtime.h"
#include <hip/driver_types.h>
#include <gsl/gsl_matrix.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include "WithCuda.h"

namespace withcuda {

void cuda_assert(hipError_t code, const char* file = __FILE__, int line = __LINE__)
{
    if (code != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(code) << " at " << file << ":" << line << std::endl;
        exit(code);
    }
}

__global__ void gain_compute_phase_kernel(const double* k_du_sin_cos, const double* k_du_sin_sin, double n, double m, double alpha, double PHI, double* phase_real, double* phase_img, size_t size)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        double tmp_img = -((-n * k_du_sin_cos[i]) + (-m * k_du_sin_sin[i]) + alpha + PHI);
        phase_real[i] += cos(tmp_img);
        phase_img[i] += sin(tmp_img);
    }
}

void gain_compute_phase(int max_threads_per_block, const cuda_matrix& k_du_sin_cos, const cuda_matrix& k_du_sin_sin, double n, double m, double alpha, double PHI, cuda_cmatrix& phase)
{
    int num_blocks = (k_du_sin_cos.rows * k_du_sin_cos.cols + max_threads_per_block - 1) / max_threads_per_block;
    gain_compute_phase_kernel<<<num_blocks, max_threads_per_block>>>(k_du_sin_cos.data, k_du_sin_sin.data, n, m, alpha, PHI, phase.data_real, phase.data_img, phase.rows * phase.cols);
}

void gsl_matrix_to_cuda_matrix(cuda_matrix& cuda, const gsl_matrix* gsl)
{
    cuda.rows = gsl->size1;
    cuda.cols = gsl->size2;
    cuda_assert(hipMalloc(&cuda.data, cuda.rows * cuda.cols * sizeof(double)), __FILE__, __LINE__);
    cuda_assert(hipMemcpy(cuda.data, gsl->data, cuda.rows * cuda.cols * sizeof(double), hipMemcpyHostToDevice), __FILE__, __LINE__);
}

void cuda_matrix_to_gsl_matrix(gsl_matrix* gsl, const cuda_matrix& cuda)
{
    cuda_assert(hipMemcpy(gsl->data, cuda.data, cuda.rows * cuda.cols * sizeof(double), hipMemcpyDeviceToHost), __FILE__, __LINE__);
}

void cuda_cmatrix_to_gsl_cmatrix(gsl_matrix_complex* gsl, const cuda_cmatrix& cuda)
{
    double* data_real = (double*) malloc(cuda.rows * cuda.cols * sizeof(double));
    double* data_img = (double*) malloc(cuda.rows * cuda.cols * sizeof(double));
    cuda_assert(hipMemcpy(data_real, cuda.data_real, cuda.rows * cuda.cols * sizeof(double), hipMemcpyDeviceToHost), __FILE__, __LINE__);
    cuda_assert(hipMemcpy(data_img, cuda.data_img, cuda.rows * cuda.cols * sizeof(double), hipMemcpyDeviceToHost), __FILE__, __LINE__);
    for (unsigned int i = 0; i < cuda.rows; i++)
        for (unsigned int j = 0; j < cuda.cols; j++)
            gsl_matrix_complex_set(gsl, i, j, {data_real[i * cuda.cols + j], data_img[i * cuda.cols + j]});
    free(data_real);
    free(data_img);
}

void cuda_matrix_alloc(cuda_matrix& cuda, unsigned int rows, unsigned int cols)
{
    cuda.rows = rows;
    cuda.cols = cols;
    cuda_assert(hipMalloc(&cuda.data, rows * cols * sizeof(double)), __FILE__, __LINE__);
    cuda_assert(hipMemset(cuda.data, 0, rows * cols * sizeof(double)), __FILE__, __LINE__);
}

void cuda_cmatrix_alloc(cuda_cmatrix& cuda, unsigned int rows, unsigned int cols)
{
    cuda.rows = rows;
    cuda.cols = cols;
    cuda_assert(hipMalloc(&cuda.data_real, rows * cols * sizeof(double)), __FILE__, __LINE__);
    cuda_assert(hipMalloc(&cuda.data_img, rows * cols * sizeof(double)), __FILE__, __LINE__);
    cuda_assert(hipMemset(cuda.data_real, 0, rows * cols * sizeof(double)), __FILE__, __LINE__);
    cuda_assert(hipMemset(cuda.data_img, 0, rows * cols * sizeof(double)), __FILE__, __LINE__);
}

void cuda_matrix_free(cuda_matrix& cuda)
{
    cuda_assert(hipFree(cuda.data), __FILE__, __LINE__);
    cuda.rows = 0;
    cuda.cols = 0;
    cuda.data = nullptr;
}

void cuda_cmatrix_free(cuda_cmatrix& cuda)
{
    cuda_assert(hipFree(cuda.data_real), __FILE__, __LINE__);
    cuda_assert(hipFree(cuda.data_img), __FILE__, __LINE__);
    cuda.rows = 0;
    cuda.cols = 0;
    cuda.data_real = nullptr;
    cuda.data_img = nullptr;
}

void set_cuda_device(int device)
{
    if (device < 0)
        throw std::runtime_error("Invalid cuda device id, must be >= 0");
    cuda_assert(hipSetDevice(device), __FILE__, __LINE__);
}

int get_cuda_max_threads_per_block()
{
    int device = hipGetDevice(&device);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);

    return prop.maxThreadsPerBlock;
}

} // namespace withcuda
